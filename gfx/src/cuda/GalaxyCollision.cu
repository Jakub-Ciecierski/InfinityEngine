#include "hip/hip_runtime.h"
#include "GalaxyCollision.cuh"

__device__ int doPrint = 0;

__device__
float3 bodiesInteraction2(float4 body1, float4 body2, float3 acceleration)
{
    float EPS2 = 0.1f;
    float3 r;

    r.x = body2.x - body1.x;
    r.y = body2.y - body1.y;
    r.z = body2.z - body1.z;

    // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]  
    float distSqr = sqrtf(r.x * r.x + r.y * r.y + r.z * r.z);
    distSqr *= distSqr;
    distSqr += EPS2;

    // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]  
    float distSixth = distSqr * distSqr * distSqr;
    float invDistCube = 1.0f / sqrtf(distSixth);

    // s = m_j * invDistCube [1 FLOP]  
    float s = body2.w * invDistCube;

    acceleration.x += r.x * s;
    acceleration.y += r.y * s;
    acceleration.z += r.z * s;

    return acceleration;
}

__device__
float3 bodiesInteraction(float4 body1, float4 body2, float3 acceleration)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    float EPS2 = 0.01f;
    float ep = 1.0f;

    float3 r;
    r.x = body2.x - body1.x;
    r.y = body2.y - body1.y;
    r.z = body2.z - body1.z;

    float distSqr = (r.x * r.x) + (r.y * r.y) + (r.z * r.z);
    //distSqr *= distSqr;
    distSqr += EPS2;

    float dist = sqrtf(distSqr);
    float distCube = dist * dist * dist;

    float s = (body2.w) / distCube;

    acceleration.x += r.x * s * ep;
    acceleration.y += r.y * s * ep;
    acceleration.z += r.z * s * ep;
/*
    if(tid == 0 && r.y > 10.0)
    {
        doPrint = 99;
        printf("\n");
        
        printf("body1.w: %lf \n", body1.w);
        printf("body2.w: %lf \n", body2.w);
        
        printf("\n");
        
        printf("r.x: %lf \n", r.x);
        printf("r.y: %lf \n", r.y);
        printf("r.z: %lf \n", r.z);
        
        printf("\n");
        
        printf("distSqr: %lf \n", distSqr);
        printf("dist: %lf \n", dist);
        printf("distCube: %lf \n", distCube);
        printf("s: %lf \n", s);
        
        printf("\n");
        
        printf("acc.x: %lf \n", acceleration.x);
        printf("acc.y: %lf \n", acceleration.y);
        printf("acc.z: %lf \n", acceleration.z);
    }
*/
    return acceleration;
}

__device__ 
float3 tileAcceleration(float4 currPosition, float3 acceleration)
{
    int i;
    extern __shared__ float4 shPosition[];
    for (i = 0; i < blockDim.x; i++) {
        acceleration = bodiesInteraction(currPosition, shPosition[i], acceleration);
    }
    return acceleration;
}

__global__
void calculateForcesKernel(float4* bodyDescription, float3* acceleration, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size)
    {
        extern __shared__ float4 shPosition[];

        float4 currPosition;
        float3 acc = { 0.0f, 0.0f, 0.0f };

        int i, tile;

        currPosition = bodyDescription[tid];
        for (i = 0, tile = 0; i < size; i += blockDim.x, tile++)
        {
            int idx = tile * blockDim.x + threadIdx.x;
            if (idx < size){
                shPosition[threadIdx.x] = bodyDescription[idx];
                __syncthreads();
                acc = tileAcceleration(currPosition, acc);
                __syncthreads();
            }
            
        }

        // Save the result in global memory for the integration step.  
        acceleration[tid] = acc;
    }
}

__host__
void galaxyCollisionLogic(float4* d_bodyDescription, float3* d_acceleration, int size)
{
    double4 test;
    dim3 DimGrid((size / MAX_BLOCK_THREAD_COUNT) + 1, 1, 1);
    dim3 DimBlock(MAX_BLOCK_THREAD_COUNT, 1, 1);

    //printf("DimGrid: x: %d, y: %d, z: %d \n", DimGrid.x, DimGrid.y, DimGrid.z);
    //printf("DimBlock: x: %d, y: %d, z: %d \n\n", DimBlock.x, DimBlock.y, DimBlock.z);

    calculateForcesKernel << <DimGrid, DimBlock, MAX_BLOCK_THREAD_COUNT * sizeof(float4)>> >
        (d_bodyDescription, d_acceleration, size);

    hipDeviceSynchronize();

    doPrint = 0;
}

__host__
void galaxyCollisionInit(float4* bodyDescription, float3* acceleration, int count)
{
    float4* d_bodyDescription;
    float3* d_acceleration;

    int sizef3 = count * sizeof(float3);
    int sizef4 = count * sizeof(float4);

    hipError_t err;

    /*********** MEMORY ALLOCATION ***********/
    if ((err = hipMalloc((void**)&d_bodyDescription, sizef4)) != hipSuccess) C_ERR(err);
    if ((err = hipMalloc((void**)&d_acceleration, sizef3)) != hipSuccess) C_ERR(err);

    /*********** COPY MEMORY TO DEVICE ***********/
    if ((err = hipMemcpy(d_bodyDescription, bodyDescription, sizef4, hipMemcpyHostToDevice)) != hipSuccess) C_ERR(err);
    //if ((err = hipMemcpy(d_acceleration, acceleration, sizef3, hipMemcpyHostToDevice)) != hipSuccess) ERR(err);

    // Work
    galaxyCollisionLogic(d_bodyDescription, d_acceleration, count);

    /*********** COPY MEMORY BACK TO HOST ***********/
    if ((err = hipMemcpy(acceleration, d_acceleration, sizef3, hipMemcpyDeviceToHost)) != hipSuccess) C_ERR(err);

    /*********** FREE MEMORY ***********/
    if ((err = hipFree(d_bodyDescription)) != hipSuccess) C_ERR(err);
    if ((err = hipFree(d_acceleration)) != hipSuccess) C_ERR(err);

    // Update galaxies ...
    /*
    for (int i = 0; i < count; i+=100)
    {
        printf("Acc: %lf, %lf, %lf\n", acceleration[i].x, acceleration[i].y, acceleration[i].z);
    }*/

    // Init the body description - position and mass
}
