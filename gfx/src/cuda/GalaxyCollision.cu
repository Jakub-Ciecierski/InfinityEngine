#include "hip/hip_runtime.h"
#include "GalaxyCollision.cuh"

__device__
float3 bodiesInteraction(float4 body1, float4 body2, float3 acceleration)
{
    float EPS2 = 0.0001f;
    float3 r;

    r.x = body2.x - body1.x;
    r.y = body2.y - body1.y;
    r.z = body2.z - body1.z;

    // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]  
    float distSqr = sqrtf(r.x * r.x + r.y * r.y + r.z * r.z);
    distSqr *= distSqr;
    distSqr += EPS2;

    // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]  
    float distSixth = distSqr * distSqr * distSqr;
    float invDistCube = 1.0f / sqrtf(distSixth);

    // s = m_j * invDistCube [1 FLOP]  
    float s = body2.w * invDistCube;

    acceleration.x += r.x * s;
    acceleration.y += r.y * s;
    acceleration.z += r.z * s;

    return acceleration;
}

__device__
float3 bodiesInteraction2(float4 body1, float4 body2, float3 acceleration)
{
    float EPS2 = 0.0001f;

    float3 r;
    r.x = body2.x - body1.x;
    r.y = body2.y - body1.y;
    r.z = body2.z - body1.z;

    float distSqr = sqrtf(r.x * r.x + r.y * r.y + r.z * r.z);
    distSqr *= distSqr;
    distSqr += EPS2;

    float dist = sqrtf(distSqr);
    float distCube = dist * dist * dist;

    float s = body2.w / distCube;

    acceleration.x += r.x * s;
    acceleration.y += r.y * s;
    acceleration.z += r.z * s;

    return acceleration;
}

__device__ 
float3 tileAcceleration(float4 currPosition, float3 acceleration)
{
    int i;
    extern __shared__ float4 shPosition[];
    for (i = 0; i < blockDim.x; i++) {
        acceleration = bodiesInteraction(currPosition, shPosition[i], acceleration);
    }
    return acceleration;
}

__global__
void calculateForcesKernel(float4* bodyDescription, float3* acceleration, int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size)
    {
        extern __shared__ float4 shPosition[];

        float4 currPosition;
        float3 acc = { 0.0f, 0.0f, 0.0f };

        int i, tile;

        currPosition = bodyDescription[tid];
        for (i = 0, tile = 0; i < size; i += blockDim.x, tile++)
        {
            int idx = tile * blockDim.x + threadIdx.x;
            if (idx < size){
                shPosition[threadIdx.x] = bodyDescription[idx];
                __syncthreads();
                acc = tileAcceleration(currPosition, acc);
                __syncthreads();
            }
            
        }

        // Save the result in global memory for the integration step.  
        acceleration[tid] = acc;
    }
}

__host__
void galaxyCollisionLogic(float4* d_bodyDescription, float3* d_acceleration, int size)
{
    dim3 DimGrid((size / MAX_BLOCK_THREAD_COUNT) + 1, 1, 1);
    dim3 DimBlock(MAX_BLOCK_THREAD_COUNT, 1, 1);

    //printf("DimGrid: x: %d, y: %d, z: %d \n", DimGrid.x, DimGrid.y, DimGrid.z);
    //printf("DimBlock: x: %d, y: %d, z: %d \n\n", DimBlock.x, DimBlock.y, DimBlock.z);

    calculateForcesKernel << <DimGrid, DimBlock, MAX_BLOCK_THREAD_COUNT * sizeof(float4)>> >
        (d_bodyDescription, d_acceleration, size);

    hipDeviceSynchronize();
}

__host__
void galaxyCollisionInit(Star*** galaxies, int galaxyCount, int n)
{
    int count = n * galaxyCount;

    int sizef3 = count * sizeof(float3);
    int sizef4 = count * sizeof(float4);

    //fprintf(stderr, "[Cuda] Bodies Count: %d \n", count);

    float4* bodyDescription = (float4*)malloc(sizef4);
    float3* acceleration = (float3*)malloc(sizef3);

    // Init the body description - position and mass
    for (int i = 0; i < galaxyCount; i++)
    {
        for (int j = 0; j < n; j++)
        {
            Star* star = galaxies[i][j];

            bodyDescription[i*n + j].x = star->x;
            bodyDescription[i*n + j].y = star->y;
            bodyDescription[i*n + j].z = star->z;
            bodyDescription[i*n + j].w = star->mass;
        }
    }

    float4* d_bodyDescription;
    float3* d_acceleration;

    hipError_t err;

    /*********** MEMORY ALLOCATION ***********/
    if ((err = hipMalloc((void**)&d_bodyDescription, sizef4)) != hipSuccess) ERR(err);
    if ((err = hipMalloc((void**)&d_acceleration, sizef3)) != hipSuccess) ERR(err);

    /*********** COPY MEMORY TO DEVICE ***********/
    if ((err = hipMemcpy(d_bodyDescription, bodyDescription, sizef4, hipMemcpyHostToDevice)) != hipSuccess) ERR(err);
    //if ((err = hipMemcpy(d_acceleration, acceleration, sizef3, hipMemcpyHostToDevice)) != hipSuccess) ERR(err);

    // Work
    galaxyCollisionLogic(d_bodyDescription, d_acceleration, count);

    /*********** COPY MEMORY BACK TO HOST ***********/
    if ((err = hipMemcpy(acceleration, d_acceleration, sizef3, hipMemcpyDeviceToHost)) != hipSuccess) ERR(err);

    /*********** FREE MEMORY ***********/
    if ((err = hipFree(d_bodyDescription)) != hipSuccess) ERR(err);
    if ((err = hipFree(d_acceleration)) != hipSuccess) ERR(err);

    // Update galaxies ...
    /*
    for (int i = 0; i < count; i+=100)
    {
        printf("Acc: %lf, %lf, %lf\n", acceleration[i].x, acceleration[i].y, acceleration[i].z);
    }*/

    for (int i = 0; i < galaxyCount; i++)
    {
        for (int j = 0; j < n; j++)
        {
            Star* star = galaxies[i][j];

            float3 acc = acceleration[i*n + j];
            star->Update(acc.x, acc.y, acc.z);
        }
    }

    free(bodyDescription);
    free(acceleration);
}